#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
#include <memory>
#include <string>
#include "hip/hip_runtime.h"
#include "FileOperations.h"
#include "NewtonSolverGPUFunctions.h"
#include <EditionalTools.h>
#include "NewtonSolverCUDA.h"
#include <chrono>

NewtonSolverCUDA::NewtonSolverCUDA(DataInitializerCUDA* dataInitializer,
        const Settings::SettingsData& settings, SystemInfo& sinfo)
    : settings_{settings}
    , sinfo_{sinfo}
{
	data = dataInitializer;
}

NewtonSolverCUDA::~NewtonSolverCUDA() {
}
void NewtonSolverCUDA::gpu_cublasInverse(DataInitializerCUDA* data) {
    hipblasStatus_t s1 = hipblasDgetrfBatched(
                data->cublasContextHandler, data->MATRIX_SIZE,
                data->cublas_ajacobian_d, data->MATRIX_SIZE,
                nullptr, //data->cublas_pivot,
                data->cublas_info, 1);

    const double alpha = 1.0;
    hipblasStatus_t s2 = hipblasDtrsm(
                data->cublasContextHandler, HIPBLAS_SIDE_LEFT,
                HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT,
                data->MATRIX_SIZE, 1, &alpha,
                data->jacobian_d, data->MATRIX_SIZE,
                data->funcs_value_d, data->MATRIX_SIZE);

    hipblasStatus_t s3 = hipblasDtrsm(
                data->cublasContextHandler, HIPBLAS_SIDE_LEFT,
                HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                data->MATRIX_SIZE, 1, &alpha,
                data->jacobian_d, data->MATRIX_SIZE,
                data->funcs_value_d, data->MATRIX_SIZE);

    std::cout << "TRF/TRS status: " << s1 << " " << s2 << " " << s3 << std::endl;
}

void NewtonSolverCUDA::gpu_newton_solve() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int version = prop.major;
    std::unique_ptr<FileOperations> file_op = std::make_unique<FileOperations>(settings_.path);
    std::string file_name = "gpu_newton_solver_" + std::to_string(data->file_name) + ".csv";
    file_op->create_file(file_name, 4);
    file_op->append_file_headers(data->csv_header);

    NewtonSolverGPUFunctions::gpu_dummy_warmup << <1, 32 >> > ();
    hipDeviceSynchronize();
    std::cout << "GPU Newton solver\n";
    int x_blocks_count = (data->MATRIX_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int iterations_count = 0;
    double dx = 0;

    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(x_blocks_count, data->MATRIX_SIZE, 1);

    double* delta = new double[data->MATRIX_SIZE];

	auto start_total = std::chrono::steady_clock::now();

    hipMemcpy(data->points_d, data->points_h, data->MATRIX_SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(data->indexes_d, data->indexes_h, data->MATRIX_SIZE * data->MATRIX_SIZE * sizeof(double), hipMemcpyHostToDevice);

    hipStream_t myStream;
    hipStreamCreate(&myStream);

    do {
        iterations_count++;

#ifdef INTERMEDIATE_RESULTS
		auto start = std::chrono::steady_clock::now();
#endif
		std::cout << "Power: " << data->equation->get_power() << "\n";
        NewtonSolverGPUFunctions::gpu_compute_func_values << <gridDim, blockDim, blockDim.x * sizeof(double) >> > (
            data->points_d, data->indexes_d, data->intermediate_funcs_value_d, data->MATRIX_SIZE, version, data->equation->get_power());
        hipDeviceSynchronize();

        hipMemcpy(data->intermediate_funcs_value_h, data->intermediate_funcs_value_d, x_blocks_count * data->MATRIX_SIZE * sizeof(double), hipMemcpyDeviceToHost);

        for (int i = 0; i < data->MATRIX_SIZE; i++) {
            data->funcs_value_h[i] = -data->vector_b_h[i];
            for (int j = 0; j < x_blocks_count; j++) {
                data->funcs_value_h[i] += data->intermediate_funcs_value_h[i * x_blocks_count + j];
            }
        }
        hipMemcpy(data->funcs_value_d, data->funcs_value_h, data->MATRIX_SIZE * sizeof(double), hipMemcpyHostToDevice);

#ifdef INTERMEDIATE_RESULTS
		auto end = std::chrono::steady_clock::now();
        data->intermediate_results[0] = std::chrono::duration<double>(end - start).count();
		start = std::chrono::steady_clock::now();
#endif

        NewtonSolverGPUFunctions::gpu_compute_jacobian << <gridDim, blockDim >> > (
            data->points_d, data->indexes_d, data->jacobian_d, data->MATRIX_SIZE, data->equation->get_power());
        hipDeviceSynchronize();

        //hipMemcpy(data->jacobian_h, data->jacobian_d, data->MATRIX_SIZE * data->MATRIX_SIZE * sizeof(double), hipMemcpyDeviceToHost);

#ifdef INTERMEDIATE_RESULTS
        end = std::chrono::steady_clock::now();
        data->intermediate_results[1] = std::chrono::duration<double>(end - start).count();
        start = std::chrono::steady_clock::now();
#endif
        gpu_cublasInverse(data);
        hipDeviceSynchronize();
#ifdef INTERMEDIATE_RESULTS
        end = std::chrono::steady_clock::now();
        data->intermediate_results[2] = std::chrono::duration<double>(end - start).count();
        start = std::chrono::steady_clock::now();
#endif

        hipMemcpy(data->funcs_value_h, data->funcs_value_d, data->MATRIX_SIZE * sizeof(double), hipMemcpyDeviceToHost);
        dx = 0.0;
        for (size_t i = 0; i < data->MATRIX_SIZE; ++i) {
            data->points_h[i] -= data->funcs_value_h[i];
            dx = std::max(dx, std::abs(data->funcs_value_h[i]));
        }

        hipMemcpy(data->points_d, data->points_h, data->MATRIX_SIZE * sizeof(double), hipMemcpyHostToDevice);

#ifdef INTERMEDIATE_RESULTS
        end = std::chrono::steady_clock::now();
        data->intermediate_results[3] = std::chrono::duration<double>(end - start).count();

        tools::print_intermediate_result(data, iterations_count, dx);
#endif
        file_op->append_file_data(
                    data->intermediate_results, data->MATRIX_SIZE,
                    data->nnz_row, iterations_count,
                    sinfo_.mem_rss_usage_get(), sinfo_.gpu_mem_usage_get(),
                    "cuBLAS", data->settings.label);
    } while (dx > TOLERANCE);
	file_op->close_file();

	auto end_total = std::chrono::steady_clock::now();
    data->total_elapsed_time = std::chrono::duration<double>(end_total - start_total).count();

    tools::print_solution(data, iterations_count);
    hipStreamDestroy(myStream);
    delete[] delta;
}
